#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2023, Inria
 * GRAPHDECO research group, https://team.inria.fr/graphdeco
 * All rights reserved.
 *
 * This software is free for non-commercial, research and evaluation use 
 * under the terms of the LICENSE.md file.
 *
 * For inquiries contact  george.drettakis@inria.fr
 */
#include <stdio.h>
#include "rasterizer_impl.h"
#include <iostream>
#include <fstream>
#include <algorithm>
#include <numeric>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""
#include <hipcub/hipcub.hpp>
#include <cub/device/device_radix_sort.cuh>
#define GLM_FORCE_CUDA
#include <glm/glm.hpp>

#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
namespace cg = cooperative_groups;

#include "auxiliary.h"
#include "forward.h"
#include "backward.h"

// Helper function to find the next-highest bit of the MSB
// on the CPU.
uint32_t getHigherMsb(uint32_t n)
{
	uint32_t msb = sizeof(n) * 4;
	uint32_t step = msb;
	while (step > 1)
	{
		step /= 2;
		if (n >> msb)
			msb += step;
		else
			msb -= step;
	}
	if (n >> msb)
		msb++;
	return msb;
}

// Wrapper method to call auxiliary coarse frustum containment test.
// Mark all Primitives that pass it.
__global__ void checkFrustum(int P,
	const float* orig_points,
	const float* viewmatrix,
	const float* projmatrix,
	bool* present)
{
	auto idx = cg::this_grid().thread_rank();
	if (idx >= P)
		return;
	float3 p_view;
	present[idx] = in_frustum(idx, orig_points, viewmatrix, projmatrix, false, p_view);
}


// Generates one key/value pair for all Primitive / tile overlaps. 
// Run once per primitive (1:N mapping).
__global__ void duplicateWithKeys(
	int P,
	const float2* points_xy,
	const float* depths,
	const uint32_t* offsets,
	uint64_t* primitive_keys_unsorted,
	uint32_t* primitive_values_unsorted,
	int* radii,
	dim3 grid,
	const float* transMats,
	const glm::vec2* scales,
	const float lambda)
{
	auto idx = cg::this_grid().thread_rank();
	if (idx >= P)
		return;

	float3 *T_ = (float3*)transMats;

	const float3 Tu = {T_[idx * 3 + 0].x, T_[idx * 3 + 0].y, T_[idx * 3 + 0].z};
	const float3 Tv = {T_[idx * 3 + 1].x, T_[idx * 3 + 1].y, T_[idx * 3 + 1].z};
	const float3 Tw = {T_[idx * 3 + 2].x, T_[idx * 3 + 2].y, T_[idx * 3 + 2].z};
	const float2 Tscale = {scales[idx].x, scales[idx].y};

	// Generate no key/value pair for invisible primitives
	if (radii[idx] > 0)
	{
		// Find this primitive's offset in buffer for writing keys/values.
		uint32_t off = (idx == 0) ? 0 : offsets[idx - 1];
		uint2 rect_min, rect_max;

		getRect(points_xy[idx], radii[idx], rect_min, rect_max, grid);
		
		// For each tile that the bounding rect overlaps, emit a 
		// key/value pair. The key is |  tile ID  |      depth      |,
		// and the value is the ID of the primitive. Sorting the values 
		// with this key yields primitive IDs in a list, such that they
		// are first sorted by tile and then by depth. 
		for (int y = rect_min.y; y < rect_max.y; y++)
		{
			for (int x = rect_min.x; x < rect_max.x; x++)
			{
				uint64_t key = y * grid.x + x;

				float pix_y = (float)y * BLOCK_Y + BLOCK_Y / 2.0f;
				float pix_x = (float)x * BLOCK_X + BLOCK_X / 2.0f;
				float2 pixf = { (float)pix_x, (float)pix_y};

				float3 k = pix_x * Tw - Tu;
				float3 l = pix_y * Tw - Tv;
				float3 p = cross(k, l);
				float2 s = {p.x / p.z, p.y / p.z};  // ray-plane intersection (u,v) in local coordinate
				float depth = (s.x * Tw.x + s.y * Tw.y) + Tw.z ;

				float px = s.x * Tscale.x;
				float py = s.y * Tscale.y;
				float wx = (Tscale.x - abs(px)) * lambda;
				float awx = (wx >= 0.0f) ? 1.0f : 2.0f*(1.0f / (1.0f + exp(-wx)));
				float wy = (Tscale.y - abs(py)) * lambda;
				float awy = (wy >= 0.0f) ? 1.0f : 2.0f*(1.0f / (1.0f + exp(-wy)));
				float beta = (awx < awy) ? 1.0f : 0.0f;
				const float alpha = beta * awx + (1.0f - beta) * awy;

				if(depth < near_n || p.z >= 0.0)
				{
					depth = 20000.0f;
				}

				key <<= 32;
				key |= *((uint32_t*)&depth);

				primitive_keys_unsorted[off] = key;
				primitive_values_unsorted[off] = idx;
				off++;
			}
		}
	}
}

// Check keys to see if it is at the start/end of one tile's range in 
// the full sorted list. If yes, write start/end of this tile. 
// Run once per instanced (duplicated) primitive ID.
__global__ void identifyTileRanges(int L, uint64_t* point_list_keys, uint2* ranges)
{
	auto idx = cg::this_grid().thread_rank();
	if (idx >= L)
		return;

	// Read tile ID from key. Update start/end of tile range if at limit.
	uint64_t key = point_list_keys[idx];
	uint32_t currtile = key >> 32;
	if (idx == 0)
		ranges[currtile].x = 0;
	else
	{
		uint32_t prevtile = point_list_keys[idx - 1] >> 32;
		if (currtile != prevtile)
		{
			ranges[prevtile].y = idx;
			ranges[currtile].x = idx;
		}
	}
	if (idx == L - 1)
		ranges[currtile].y = L;
}

// Mark primitives as visible/invisible, based on view frustum testing
void CudaRasterizer::Rasterizer::markVisible(
	int P,
	float* means3D,
	float* viewmatrix,
	float* projmatrix,
	bool* present)
{
	checkFrustum << <(P + 255) / 256, 256 >> > (
		P,
		means3D,
		viewmatrix, projmatrix,
		present);
}

CudaRasterizer::GeometryState CudaRasterizer::GeometryState::fromChunk(char*& chunk, size_t P)
{
	GeometryState geom;
	obtain(chunk, geom.depths, P, 128);
	obtain(chunk, geom.clamped, P * 3, 128);
	obtain(chunk, geom.internal_radii, P, 128);
	obtain(chunk, geom.means2D, P, 128);
	obtain(chunk, geom.transMat, P * 9, 128);
	obtain(chunk, geom.normal_opacity, P, 128);
	obtain(chunk, geom.rgb, P * 3, 128);
	obtain(chunk, geom.tiles_touched, P, 128);
	hipcub::DeviceScan::InclusiveSum(nullptr, geom.scan_size, geom.tiles_touched, geom.tiles_touched, P);
	obtain(chunk, geom.scanning_space, geom.scan_size, 128);
	obtain(chunk, geom.point_offsets, P, 128);
	return geom;
}

CudaRasterizer::ImageState CudaRasterizer::ImageState::fromChunk(char*& chunk, size_t N)
{
	ImageState img;
	obtain(chunk, img.accum_alpha, N * 4, 128);
	obtain(chunk, img.n_contrib, N * (2+1+30), 128);
	obtain(chunk, img.ranges, N, 128);
	return img;
}

CudaRasterizer::BinningState CudaRasterizer::BinningState::fromChunk(char*& chunk, size_t P)
{
	BinningState binning;
	obtain(chunk, binning.point_list, P, 128);
	obtain(chunk, binning.point_list_unsorted, P, 128);
	obtain(chunk, binning.point_list_keys, P, 128);
	obtain(chunk, binning.point_list_keys_unsorted, P, 128);
	hipcub::DeviceRadixSort::SortPairs(
		nullptr, binning.sorting_size,
		binning.point_list_keys_unsorted, binning.point_list_keys,
		binning.point_list_unsorted, binning.point_list, P);
	obtain(chunk, binning.list_sorting_space, binning.sorting_size, 128);
	return binning;
}

// Forward rendering procedure for differentiable rasterization
// of primitives.
int CudaRasterizer::Rasterizer::forward(
	std::function<char* (size_t)> geometryBuffer,
	std::function<char* (size_t)> binningBuffer,
	std::function<char* (size_t)> imageBuffer,
	const int P, int D, int M,
	const float* background,
	const int width, int height,
	const float* means3D,
	const float* shs,
	const float* colors_precomp,
	const float* opacities,
	const float* scales,
	const float scale_modifier,
	const float* rotations,
	const float* transMat_precomp,
	const float* viewmatrix,
	const float* projmatrix,
	const float* cam_pos,
	const float tan_fovx, float tan_fovy,
	const bool prefiltered,
	float* out_color,
	float* out_others,
	int* radii,
	bool debug,
	const float lambda,
	const float* image_center,
	const bool hard_render)
{
	const float focal_y = height / (2.0f * tan_fovy);
	const float focal_x = width / (2.0f * tan_fovx);

	size_t chunk_size = required<GeometryState>(P);
	char* chunkptr = geometryBuffer(chunk_size);
	GeometryState geomState = GeometryState::fromChunk(chunkptr, P);

	if (radii == nullptr)
	{
		radii = geomState.internal_radii;
	}
	
	dim3 tile_grid((width + BLOCK_X - 1) / BLOCK_X, (height + BLOCK_Y - 1) / BLOCK_Y, 1);
	dim3 block(BLOCK_X, BLOCK_Y, 1);

	// Dynamically resize image-based auxiliary buffers during training
	size_t img_chunk_size = required<ImageState>(width * height);
	char* img_chunkptr = imageBuffer(img_chunk_size);
	ImageState imgState = ImageState::fromChunk(img_chunkptr, width * height);

	if (NUM_CHANNELS != 3 && colors_precomp == nullptr)
	{
		throw std::runtime_error("For non-RGB, provide precomputed primitive colors!");
	}

	// Run preprocessing per-primitive (transformation, bounding, conversion of SHs to RGB)
	CHECK_CUDA(FORWARD::preprocess(
		P, D, M,
		means3D,
		(glm::vec2*)scales,
		scale_modifier,
		(glm::vec4*)rotations,
		opacities,
		shs,
		geomState.clamped,
		transMat_precomp,
		colors_precomp,
		viewmatrix, projmatrix,
		(glm::vec3*)cam_pos,
		width, height,
		focal_x, focal_y,
		tan_fovx, tan_fovy,
		radii,
		geomState.means2D,
		geomState.depths,
		geomState.transMat,
		geomState.rgb,
		geomState.normal_opacity,
		tile_grid,
		geomState.tiles_touched,
		prefiltered,
		image_center
	), debug)

	// Compute prefix sum over full list of touched tile counts by primitives
	// E.g., [2, 3, 0, 2, 1] -> [2, 5, 5, 7, 8]
	CHECK_CUDA(hipcub::DeviceScan::InclusiveSum(geomState.scanning_space, geomState.scan_size, geomState.tiles_touched, geomState.point_offsets, P), debug)

	// Retrieve total number of primitive instances to launch and resize aux buffers
	int num_rendered;
	CHECK_CUDA(hipMemcpy(&num_rendered, geomState.point_offsets + P - 1, sizeof(int), hipMemcpyDeviceToHost), debug);

	size_t binning_chunk_size = required<BinningState>(num_rendered);
	char* binning_chunkptr = binningBuffer(binning_chunk_size);
	BinningState binningState = BinningState::fromChunk(binning_chunkptr, num_rendered);

	// For each instance to be rendered, produce adequate [ tile | depth ] key 
	// and corresponding dublicated primitive indices to be sorted
	dim3 tile_grid_tmp((width + BLOCK_X - 1) / BLOCK_X, (height + BLOCK_Y - 1) / BLOCK_Y, 1);
	dim3 block_tmp(BLOCK_X, BLOCK_Y, 1);
	duplicateWithKeys << <(P + 255) / 256, 256 >> > (
		P,
		geomState.means2D,
		geomState.depths,
		geomState.point_offsets,
		binningState.point_list_keys_unsorted,
		binningState.point_list_unsorted,
		radii,
		tile_grid_tmp,
		geomState.transMat,
		(glm::vec2*)scales,
		lambda)

	CHECK_CUDA(, debug)

	int bit = getHigherMsb(tile_grid.x * tile_grid.y);

	// Sort complete list of (duplicated) primitive indices by keys
	CHECK_CUDA(hipcub::DeviceRadixSort::SortPairs(
		binningState.list_sorting_space,
		binningState.sorting_size,
		binningState.point_list_keys_unsorted, binningState.point_list_keys,
		binningState.point_list_unsorted, binningState.point_list,
		num_rendered, 0, 32 + bit), debug)

	CHECK_CUDA(hipMemset(imgState.ranges, 0, tile_grid.x * tile_grid.y * sizeof(uint2)), debug);

	// Identify start and end of per-tile workloads in sorted list
	if (num_rendered > 0)
		identifyTileRanges << <(num_rendered + 255) / 256, 256 >> > (
			num_rendered,
			binningState.point_list_keys,
			imgState.ranges);
	CHECK_CUDA(, debug)

	// Let each tile blend its range of primitives independently in parallel
	const float* feature_ptr = colors_precomp != nullptr ? colors_precomp : geomState.rgb;
	const float* transMat_ptr = transMat_precomp != nullptr ? transMat_precomp : geomState.transMat;
	
	CHECK_CUDA(FORWARD::render(
		tile_grid, block,
		imgState.ranges,
		binningState.point_list,
		width, height,
		focal_x, focal_y,
		geomState.means2D,
		feature_ptr,
		transMat_ptr,
		geomState.depths,
		geomState.normal_opacity,
		imgState.accum_alpha,
		imgState.n_contrib,
		background,
		out_color,
		out_others,
		scales,
		lambda,
		hard_render), debug)

	return num_rendered;
}

// Produce necessary gradients for optimization, corresponding
// to forward render pass
void CudaRasterizer::Rasterizer::backward(
	const int P, int D, int M, int R,
	const float* background,
	const int width, int height,
	const float* means3D,
	const float* shs,
	const float* colors_precomp,
	const float* scales,
	const float scale_modifier,
	const float* rotations,
	const float* transMat_precomp,
	const float* viewmatrix,
	const float* projmatrix,
	const float* campos,
	const float tan_fovx, float tan_fovy,
	const int* radii,
	char* geom_buffer,
	char* binning_buffer,
	char* img_buffer,
	const float* dL_dpix,
	const float* dL_depths,
	float* dL_dmean2D,
	float* dL_dnormal,
	float* dL_dopacity,
	float* dL_dcolor,
	float* dL_dmean3D,
	float* dL_dtransMat,
	float* dL_dsh,
	float* dL_dscale,
	float* dL_drot,
	bool debug,
	const float lambda,
	float* dL_dproj,
	float* dL_dtransMat2,
	const float* image_center,
	const float* scales2)
{
	GeometryState geomState = GeometryState::fromChunk(geom_buffer, P);
	BinningState binningState = BinningState::fromChunk(binning_buffer, R);
	ImageState imgState = ImageState::fromChunk(img_buffer, width * height);

	if (radii == nullptr)
	{
		radii = geomState.internal_radii;
	}

	const float focal_y = height / (2.0f * tan_fovy);
	const float focal_x = width / (2.0f * tan_fovx);

	const dim3 tile_grid((width + BLOCK_X - 1) / BLOCK_X, (height + BLOCK_Y - 1) / BLOCK_Y, 1);
	const dim3 block(BLOCK_X, BLOCK_Y, 1);

	// Compute loss gradients w.r.t. 2D mean position, conic matrix,
	// opacity and RGB of primitives from per-pixel loss gradients.
	// If we were given precomputed colors and not SHs, use them.
	const float* color_ptr = (colors_precomp != nullptr) ? colors_precomp : geomState.rgb;
	const float* depth_ptr = geomState.depths;
	const float* transMat_ptr = (transMat_precomp != nullptr) ? transMat_precomp : geomState.transMat;

	// hipEvent_t start, stop;
    // hipEventCreate(&start);
    // hipEventCreate(&stop);
	// hipEventRecord(start);

	CHECK_CUDA(BACKWARD::render(
		P,
		tile_grid,
		block,
		imgState.ranges,
		binningState.point_list,
		width, height,
		focal_x, focal_y,
		background,
		geomState.means2D,
		geomState.normal_opacity,
		color_ptr,
		transMat_ptr,
		depth_ptr,
		imgState.accum_alpha,
		imgState.n_contrib,
		dL_dpix,
		dL_depths,
		dL_dtransMat,
		(float3*)dL_dmean2D,
		dL_dnormal,
		dL_dopacity,
		dL_dcolor,
		(glm::vec4*)dL_dscale,
		scales,
		lambda,
		(glm::vec2*)dL_dproj,
		dL_dtransMat2), debug)
	
    // hipEventRecord(stop);
    // hipEventSynchronize(stop);
    // float milliseconds = 0;
    // hipEventElapsedTime(&milliseconds, start, stop);
    // std::cout << "Kernel execution took " << milliseconds << " ms" << std::endl;
    // hipEventDestroy(start);
    // hipEventDestroy(stop);

	// Take care of the rest of preprocessing. Was the precomputed covariance
	// given to us or a scales/rot pair? If precomputed, pass that. If not,
	// use the one we computed ourselves.
	// const float* transMat_ptr = (transMat_precomp != nullptr) ? transMat_precomp : geomState.transMat;
	CHECK_CUDA(BACKWARD::preprocess(P, D, M,
		(float3*)means3D,
		radii,
		shs,
		geomState.clamped,
		(glm::vec2*)scales2,
		(glm::vec4*)rotations,
		scale_modifier,
		transMat_ptr,
		viewmatrix,
		projmatrix,
		focal_x, focal_y,
		tan_fovx, tan_fovy,
		(glm::vec3*)campos,
		(float3*)dL_dmean2D, // gradient inputs
		dL_dnormal,		     // gradient inputs
		dL_dtransMat,
		dL_dcolor,
		dL_dsh,
		(glm::vec3*)dL_dmean3D,
		(glm::vec4*)dL_drot,
		(glm::vec2*)dL_dproj,
		dL_dtransMat2,
		image_center), debug)
}